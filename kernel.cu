#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <vector>
#include <cmath>
#include <random>
#include <iostream>
#include <numeric>
#include <iomanip>

#include "kernel.cuh"


__constant__ Config globalConfig;


__global__ void Init_states(hiprandState* states, long long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    hiprand_init(seed, idx, 1000, &states[idx]);
}

__global__ void InitBitstring(hiprandState* states, bool* b) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    int startBit = idx * globalConfig.bits;
    for (int i = startBit; i < startBit + globalConfig.bits; i++)
    {
        b[i] = hiprand_uniform(&states[idx]) > 0.5f;
    }
}

__device__ void Convert(bool* bits, double* values)
{
    for (int j = 0; j < globalConfig.d; j++) {
        unsigned long long dec = 0;
        for (int i = 0; i < globalConfig.bitsPerDim; i++)
        {
            dec = (dec << 1) | bits[j * globalConfig.bitsPerDim + i];

        }
        values[j] = globalConfig.a + dec * (globalConfig.b - globalConfig.a) / ((1ull << globalConfig.bitsPerDim) - 1);
    }
}
__global__ void GenRealValues(bool* bits, double* values) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    Convert(bits + idx * globalConfig.bits, values + idx * globalConfig.d);
}

__device__ double Rastrigin(double* v, int dimensions) {

    double res = 10 * dimensions;
    for (int i = 0; i < dimensions; i++) {
        res += v[i] * v[i] - 10 * cos(2 * M_PI * v[i]);
    }
    return res;
}

__device__ double Michalewicz(double* v, int dimensions) {
    double res = 0;
    for (int i = 0; i < dimensions; i++) {
        res += sin(v[i]) * pow(sin(((i + 1) * v[i] * v[i]) / M_PI), 20);
    }
    return -res;
}

//reminder to check if this si actually dejong
__device__ double Dejong(double* v, int dimensions) {
    double res = 0;
    for (int i = 0; i < dimensions; i++) {
        res += v[i] * v[i];
    }
    return res;
}

__device__ double Schwefel(double* v, int dimensions) {
    double res = 0;
    for (int i = 0; i < dimensions; i++) {
        res += -v[i] * sin(sqrt(abs(v[i])));
    }
    return res;
}

__device__ double Eval(double* values)
{
    switch (globalConfig.func)
    {
    case function::Rastrigin:
        return Rastrigin(values, globalConfig.d);
        break;
    case function::Michalewicz:
        return Michalewicz(values, globalConfig.d);
        break;

    case function::Schwefel:
        return Schwefel(values, globalConfig.d);
        break;

    case function::Dejong:
        return Dejong(values, globalConfig.d);
        break;
    }
}

__global__ void EvalValue(double* values, double* candidates)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    candidates[idx] = Eval(idx * globalConfig.d + values);
}



__global__ void CummulativeFitness(double* fitnessscores) {
    double result;
    for (int i = 0; i < globalConfig.it; i++) {
        result += fitnessscores[i];
    }
    for (int i = 0; i < globalConfig.it; i++) {
        fitnessscores[i] = fitnessscores[i] / result;
    }
    for (int i = 1; i < globalConfig.it; i++) {
        fitnessscores[i] = fitnessscores[i] + fitnessscores[i-1];
      
    } 
  
   
}

__global__ void mutate(hiprandState* states, bool* binPopulation, double* eval) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    for (int i = 0; i < globalConfig.it; ++i) {
        double probM = hiprand_uniform_double(&states[idx]);
        if (probM <= globalConfig.mutationRate) {
           mutateInstance(states, binPopulation, idx);
         
        }
    }
}


__device__ void mutateInstance(hiprandState* states, bool* candidate, int idx) {
    bool mutated = false;
    int startBit = idx * globalConfig.bits;
    while (!mutated) {
        for (int i = startBit; i < startBit + globalConfig.bits; ++i) {
            double p = hiprand_uniform_double(&states[idx]);
            if (p < 1 / globalConfig.bits) {
                candidate[i] = !candidate[i];
                mutated = true;
                break;

            }
        }
    }
}
__device__ void EvalFitnessdevice(double* candidates, double* fitnessscores, int idx) {
    //to be changed to e^ or even have switch case
    fitnessscores[idx] = -candidates[idx];
}
__global__ void EvalFitness(double* candidates, double* fitnessscores)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    EvalFitnessdevice(candidates, fitnessscores, idx);

}



__global__  void Algorithm(bool* bitstr, double* values, double* candidates, hiprandState* states) {

}

std::vector<double> launch(const Config& config) {

    bool* bitstr;
    double* candidates;
    double* realValues;
    double* fitnessScores;
    hiprandState* states;
    std::vector<double> result(config.it);

    // Allocate device memory
    hipMalloc(&bitstr, sizeof(bool) * config.bits * config.it);
    hipMalloc(&candidates, sizeof(double) * config.it);
    hipMalloc(&fitnessScores, sizeof(double) * config.it);
    hipMalloc(&states, sizeof(hiprandState) * config.it);
    hipMalloc(&realValues, sizeof(double) * config.it * config.d);
    hipMemcpyToSymbol(HIP_SYMBOL(globalConfig), &config, sizeof(Config));


    // Launch kernel
    Init_states << < config.blocks, config.threads >> > (states, std::random_device{}());
    InitBitstring << < config.blocks, config.threads >> > (states, bitstr);
    GenRealValues << < config.blocks, config.threads >> > (bitstr, realValues);
    EvalValue << < config.blocks, config.threads >> > (realValues, candidates);
    EvalFitness << < config.blocks, config.threads >> > (candidates, fitnessScores);
    CummulativeFitness << < 1, 1 >> > (fitnessScores);

    //Algorithm << < config.blocks, config.threads >> > (bitstr, realValues, candidates, states);


    // Copy result back to host
    hipMemcpy(result.data(), fitnessScores, sizeof(double) * config.it, hipMemcpyDeviceToHost);


    // Clean up device memory
    hipFree(bitstr);
    hipFree(candidates);
    hipFree(states);
    hipFree(realValues);

    return result;
}